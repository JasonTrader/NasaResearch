
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <iomanip>
using namespace std;

#define PI 3.1415926535897932384
#define mu0 4*PI*1e-7
#define threadsPerBlock 500

__global__ void setGhostPoints(double *rod_new, double *ghost, int numBlocks, int numseg){
  int i;
  threadIdx.x == (numBlocks - 1) ? i = numseg + 1 : i = threadIdx.x * threadsPerBlock;
  ghost[threadIdx.x] = rod_new[i];
}

__global__ void init(double *rod_new, double imax, double ldr, double rlength, int numseg){
  int i = threadIdx.x + 1;
  rod_new[i] = (1-(i*i*ldr*ldr/(3*rlength*rlength)))*3*mu0*imax*i*ldr/(4*PI*rlength*rlength);
}

__global__ void run(double *ghost, double *rod_new, double aug, int numseg, int numBlocks){
  int bi = blockIdx.x;
  int ti = threadIdx.x;
  int i = bi*threadsPerBlock + ti + 1;
  int threadsNeeded;
  bi == (numBlocks - 1) ? threadsNeeded = (numseg - (bi*threadsPerBlock)) : threadsNeeded =  threadsPerBlock;
  extern __shared__ double rod_old_s[];
  double ghost_left = ghost[bi];
  double ghost_right = ghost[bi+1];
  rod_old_s[ti] = rod_new[i];
  __syncthreads();

  if(threadsNeeded == 1){
    rod_new[i] += aug*((1+(1/(2*i)))*ghost_right + (-2-(1/(i*i)))*rod_old_s[ti] + (1-(1/(2*i)))*ghost_left);
    return;
  }

  if(i==1)
    rod_new[1]+= aug*(2*rod_old_s[2] - 4*rod_old_s[1]);
  else if(ti == 0)
    rod_new[i] += aug*((1+(1/(2*i)))*rod_old_s[ti+1] + (-2-(1/(i*i)))*rod_old_s[ti] + (1-(1/(2*i)))*ghost_left);
  else if(ti == threadsNeeded)
    rod_new[i] += aug*((1+(1/(2*i)))*ghost_right + (-2-(1/(i*i)))*rod_old_s[ti] + (1-(1/(2*i)))*rod_old_s[ti-1]);
  else if(i<(numseg + 1))
    rod_new[i] += aug*((1+(1/(2*i)))*rod_old_s[i+1] + (-2-(1/(i*i)))*rod_old_s[i] + (1-(1/(2*i)))*rod_old_s[i-1]);
}

int main(){
  FILE *myfile;
  myfile = fopen("results.txt", "w");
  double imax, rlength, eta, tstep, ldr, tottime;
  int numseg;
  printf("What is your I max? ");
  scanf("%lf", &imax);
  printf("What is the length of your rod? ");
  scanf("%lf", &rlength);
  printf("What is eta? ");
  scanf("%lf", &eta);
  printf("How many segments would you like? ");
  scanf("%d", &numseg);
  ldr = rlength/(numseg+1);
  tstep = 0.25*ldr*ldr*mu0/eta;
  printf("How long would you like to run? ");
  scanf("%lf", &tottime);

  double *h_rod, *d_rod, *d_ghost;
  size_t rod_size = (numseg + 2) * sizeof(double);
  h_rod = (double*)malloc(rod_size);
  hipMalloc(&d_rod, rod_size);

  init<<<1,numseg>>>(d_rod, imax, ldr, rlength, numseg);

  int out;
  //output r values
  for(out = 0; out<numseg+1; out++){
    fprintf( myfile, "%lf ", out*ldr );
  }
  fprintf( myfile, "%lf\n", out*ldr );

  hipMemcpy(h_rod, d_rod, rod_size, hipMemcpyDeviceToHost);

  for(out = 0; out<numseg+1; out++){
    fprintf( myfile, "%lf ", *(h_rod+out) );
  }
  fprintf( myfile, "%lf\n", *(h_rod+out) );

  double aug = eta*tstep/(mu0*ldr*ldr);
  long int total_steps = tottime / tstep;
  printf("\nSteps: %ld\n", total_steps);


  clock_t begin, end;
  double time_spent;
  begin = clock();
  int numBlocks = (numseg + threadsPerBlock -1)/threadsPerBlock;
  size_t ghost_size = (numBlocks + 1) * sizeof(double);
  hipMalloc(&d_ghost, ghost_size);

  //run
  long int steps = 0;
  while(steps< total_steps){
    setGhostPoints<<<1,numBlocks>>>(d_rod, d_ghost, numBlocks, numseg);
    run<<<numBlocks, numseg, numseg*sizeof(double)>>>(d_ghost, d_rod, aug, numseg, numBlocks);
    steps++;
  }
  hipDeviceSynchronize();

  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

  hipMemcpy(h_rod, d_rod, rod_size, hipMemcpyDeviceToHost);

  for(out = 0; out<numseg+1; out++){
    fprintf( myfile, "%lf ", *(h_rod+out) );
  }
  fprintf( myfile, "%lf\n", *(h_rod+out) );

  fprintf(myfile, "STOP\n");
  fclose(myfile);

  hipFree(d_rod);
  free(h_rod);



  cout << "\n------------------------------------\nExecution took: "<<  time_spent << " sec\n";

  return 0;
}
