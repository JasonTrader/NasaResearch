
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <iomanip>
using namespace std;

#define PI 3.1415926535897932384
#define mu0 4*PI*1e-7

__global__ void init(double *rod_old, double *rod_new, double imax, double ldr, double rlength, int total_seg){
  int i = threadIdx.x;
  rod_new[i] = (1-(i*i*ldr*ldr/(3*rlength*rlength)))*3*mu0*imax*i*ldr/(4*PI*rlength*rlength);
  if(i==0 || i==total_seg-1){
    rod_old[i] = rod_new[i];
  }
}

__global__ void run(double *rod_old, double *rod_new, double aug, long int maxSteps, int rod_size){
  int i = threadIdx.x + 1;
  long int steps = 0;
  extern __shared__ double rod_new_s[];
  extern __shared__ double rod_old_s[];
  rod_new_s[i] = rod_new[i];
  __syncthreads();

  while(steps<maxSteps){
    rod_old_s[i] = rod_new_s[i];
    __syncthreads();
    if(i==1)
      rod_new_s[1]+= aug*(2*rod_old_s[2] - 4*rod_old_s[1]);
    else if(i<(rod_size - 1))
      rod_new_s[i] += aug*((1+(1/(2*i)))*rod_old_s[i+1] + (-2-(1/(i*i)))*rod_old_s[i] + (1-(1/(2*i)))*rod_old_s[i-1]);
    steps++;
    __syncthreads();
  }

  rod_new[i] = rod_new_s[i];
}

int main(){
  FILE *myfile;
  myfile = fopen("results.txt", "w");
  double imax, rlength, eta, tstep, ldr, tottime;
  int numseg;
  printf("What is your I max? ");
  scanf("%lf", &imax);
  printf("What is the length of your rod? ");
  scanf("%lf", &rlength);
  printf("What is eta? ");
  scanf("%lf", &eta);
  printf("How many segments would you like? ");
  scanf("%d", &numseg);
  ldr = rlength/(numseg+1);
  tstep = 0.25*ldr*ldr*mu0/eta;
  printf("How long would you like to run? ");
  scanf("%lf", &tottime);

  double *h_rod, *d_rod_new, *d_rod_old;
  size_t rod_size = (numseg + 2) * sizeof(double);
  h_rod = (double*)malloc(rod_size);
  hipMalloc(&d_rod_new, rod_size);
  hipMalloc(&d_rod_old, rod_size);

  init<<<1,numseg+2>>>(d_rod_old, d_rod_new, imax, ldr, rlength, numseg + 2);

  int out;
  //output r values
  for(out = 0; out<numseg+1; out++){
    fprintf( myfile, "%lf ", out*ldr );
  }
  fprintf( myfile, "%lf\n", out*ldr );

  hipMemcpy(h_rod, d_rod_new, rod_size, hipMemcpyDeviceToHost);

  for(out = 0; out<numseg+1; out++){
    fprintf( myfile, "%lf ", *(h_rod+out) );
  }
  fprintf( myfile, "%lf\n", *(h_rod+out) );

  double aug = eta*tstep/(mu0*ldr*ldr);
  long int total_steps = tottime / tstep;
  printf("\nSteps: %ld\n", total_steps);


  clock_t begin, end;
  double time_spent;
  begin = clock();

  //run
  run<<<1,numseg + 2, (numseg+2)*sizeof(double)>>>(d_rod_old, d_rod_new, aug, total_steps, numseg+2);
  hipDeviceSynchronize();

  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

  hipMemcpy(h_rod, d_rod_new, rod_size, hipMemcpyDeviceToHost);

  for(out = 0; out<numseg+1; out++){
    fprintf( myfile, "%lf ", *(h_rod+out) );
  }
  fprintf( myfile, "%lf\n", *(h_rod+out) );

  fprintf(myfile, "STOP\n");
  fclose(myfile);



  cout << "\n------------------------------------\nExecution took: "<<  time_spent << " sec\n";

  return 0;
}
