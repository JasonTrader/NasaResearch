#include <stdlib.h>
#include <stdio.h>

//Header files
#include "globals.h"
#include "LaplacianVoltage.h"
#include "MassConservation.h"
#include "MomentumConservation.h"
#include "UCopy.h"

#define U_d(i,on) U_d+((2*i+on)*(nr+1)*(nz+1))
#define U_h(i) U_h+(i*(nr+1)*(nz+1))
#define S_d(i) S_d+(i*(nr+1)*(nz+1))
#define S_h(i) S_h+(i*(nr+1)*(nz+1))

int main(){

  //-------------------------------------------------------------------------//
  //get input

  //Propellant (AMU) (currently not used)
  scanf("%*s %*s %*s %*d");

  //Mass flow rate (kg/s) (currently not used)
  scanf("%*s %*s %*s %*lf");

  //inner r (m)
  double rIn;
  scanf("%*s %*s %*s %lf", &rIn);

  //outer r (m)
  double rOut;
  scanf("%*s %*s %*s %lf", &rOut);

  //Total rlength
  double lr = rIn - rOut;

  //z length (m)
  double lz;
  scanf("%*s %*s %*s %lf", &lz);

  //number of points in r direction
  int nr;
  scanf("%*s %*s %d", &nr);

  //number of points in z direction
  int nz;
  scanf("%*s %*s %d", &nz);

  //Start time (s)
  double startTime;
  scanf("%*s %*s %*s %lf", &startTime);

  //End time (s)
  double endTime;
  scanf("%*s %*s %*s %lf", &endTime);

//----------------------------------------------------------------------------//
  //variable setup
  double dr = lr/(nr+1);
  double dz = lz/(nz+1);
  double smallest = dr;
  if(dz < dr)
    smallest = dz;
  double dt = 0.125*smallest*smallest*MU0;///eta;//to ensure stability
  //QUESTION eta?

//---------------------------------------------------------------------------//
// Memory setup

  size_t cornerGridSize = (nr+2)*(nz+2)*sizeof(double);
  dim3 cornerGridWHalosThreadDim(R_EVALS_PER_BLOCK+2, Z_EVALS_PER_BLOCK+2);//+ 2 accounts for halo points
  int cornerGridWHalosBlockR = 1 + (nr-1)/R_EVALS_PER_BLOCK;//nr = number internal r grid points
  int cornerGridWHalosBlockZ = 1 + (nz-1)/Z_EVALS_PER_BLOCK;//nz = number internal z grid points
  dim3 cornerGridWHalosBlockDim(cornerGridWHalosBlockR, cornerGridWHalosBlockZ);

  size_t centerGridSize = (nr+1)*(nz+1)*sizeof(double);
  dim3 centerGridWHalosThreadDim(R_EVALS_PER_BLOCK+2, Z_EVALS_PER_BLOCK+2);//+ 2 accounts for halo points
  int centerGridBlockR = 1 + (nr+1-1)/R_EVALS_PER_BLOCK;//nr+1 = number internal r grid points
  int centerGridBlockZ = 1 + (nz+1-1)/Z_EVALS_PER_BLOCK;//nz + 1 = number internal z grid points
  dim3 centerGridWHalosBlockDim(centerGridBlockR, centerGridBlockZ);
  dim3 centerGridNoHalosThreadDim(R_EVALS_PER_BLOCK, Z_EVALS_PER_BLOCK);
  dim3 centerGridNoHalosBlockDim(centerGridBlockR, centerGridBlockZ);


  //Voltage
  double *voltOld_d, *voltNew_d;//Device voltage grids
  hipMalloc(&voltOld_d, cornerGridSize);
  hipMalloc(&voltNew_d, cornerGridSize);
  double *volt_h;//Host voltage grid
  volt_h = (double*)malloc(cornerGridSize);
  bool *converge_d;
  size_t convergeSize = 2*cornerGridWHalosBlockR*cornerGridWHalosBlockZ*sizeof(bool);
  hipMalloc(&converge_d, convergeSize);
  bool *converge_h;
  converge_h = (bool*)malloc(convergeSize);
  double *Er_d, *Ez_d;
  hipMalloc(&Er_d,centerGridSize);
  hipMalloc(&Ez_d,centerGridSize);
  double *Er_h, *Ez_h;
  Er_h = (double*)malloc(centerGridSize);
  Ez_h = (double*)malloc(centerGridSize);

  //Conserved Quantities
  size_t uSize = 6*centerGridSize;
  double *U_d, *S_d;
  hipMalloc(&U_d,2*uSize);
  hipMalloc(&S_d,uSize);
  double *U_h, *S_h;
  U_h = (double*)malloc(centerGridSize);
  S_h = (double*)malloc(centerGridSize);


//---------------------------------------------------------------------------//

  //TODO calculate initial conserved quantities

  //TODO calculate secondary initial quantities


  //Time loop
  double t = startTime;
  while(t < endTime){


//---------------------------------------------------------------------------//
    //Update Voltage
    getNewVoltage(cornerGridSize,convergeSize,voltOld_d,voltNew_d,volt_h,cornerGridWHalosBlockDim,
      cornerGridWHalosThreadDim,converge_d,converge_h,nr,nz,dr,dz,
      cornerGridWHalosBlockR,cornerGridWHalosBlockZ);

//---------------------------------------------------------------------------//

    getMass(U_d(massP,o),U_d(massP,n),U_d(momentumPR,o),U_d(momentumPZ,o),S_d(massP),
      U_d(massN,o),U_d(massN,n),U_d(momentumNR,o),U_d(momentumNZ,o),S_d(massN),
      nr,nz,dr,dz,dt,centerGridWHalosBlockDim,centerGridWHalosThreadDim,centerGridSize);

    getMomentum(U_d(massP,o), U_d(momentumPR,n), U_d(momentumPZ,n), U_d(momentumPR,o), U_d(momentumPZ,o), S_d(momentumPR), S_d(momentumPZ),
      U_d(massN,o), U_d(momentumNR,n), U_d(momentumNZ,n), U_d(momentumNR,o), U_d(momentumNZ,o), S_d(momentumNR), S_d(momentumNZ),
      nr,nz,dr,dz,dt,centerGridWHalosBlockDim,centerGridWHalosThreadDim,centerGridSize);

    UCopy(U_d(massP,o),U_d(massP,n), U_d(massN,o), U_d(massN,n),
      U_d(momentumPR,o), U_d(momentumPR,n), U_d(momentumNR,o), U_d(momentumNR,n),
      U_d(momentumPZ,o), U_d(momentumPZ,n), U_d(momentumNZ,o), U_d(momentumNZ,n), centerGridSize);

    //TODO Source/Sink

    t+=dt;//update time
  }

  //TODO Output results

//---------------------------------------------------------------------------//
//Free memory
  hipFree(voltOld_d);
  hipFree(voltNew_d);
  free(volt_h);
  hipFree(Er_d);
  hipFree(Ez_d);
  free(Er_h);
  free(Ez_h);
  hipFree(U_d);
  hipFree(S_d);
  free(U_h);
  free(S_h);

  return 0;
}
